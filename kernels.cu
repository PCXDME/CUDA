#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdbool.h>

#define TILE_SIZE 512
#define WARP_SIZE 32


extern "C" void CSRmatvecmult(int* start, int* J, float* Val, int N, int nnz, float* x, float *y, bool bVectorized);
extern "C" void ELLmatvecmult(int N, int num_cols_per_row , int * indices, float * data , float * x , float * y);

/**
 * Custom CUDA error check wrapper.
 */
#define checkCUDAError() do {                           \
 hipError_t error = hipGetLastError();               \
 if (error != hipSuccess) {                            \
   printf("(CUDA) %s", hipGetErrorString(error)); \
   printf(" (" __FILE__ ":%d)\n", __LINE__);  \
  }\
} while (0)

/**
 * Cuda kernel for: CSR_s(A)x = y
 */
__global__ void k_csr_mat_vec_mm(int *start, int* j, float *a_content, int num_rows, float *x, float* y) {
    // TODO: implement the scalar crs kernel
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	float result = 0.0f;

	for(int i=start[row];i<start[row+1];i++) {
		int value = a_content[i];
		int column = j[i];

		result += x[column] * value;
	}

	y[row] += result;
}

/**
 * Cuda kernel for: CSR_v(A)x = y
 */
__global__ void k_csr2_mat_vec_mm(int *start, int* j, float *a_content, int num_rows, float *x, float* y) {
	//TODO: implement the vectorized csr kernel
}

/**
 * Cuda kernel for: ELL(A)x = y
 */
__global__ void k_ell_mat_vec_mm ( int N, int num_cols_per_row , int * indices,
									float * data , float * x , float * y ) {
	//NYI: ellpack kernel
}

/**
 * Perform: CSR(A)x = y
 */
void CSRmatvecmult(int* start, int* J, float* Val, int N, int nnz, float* x, float *y, bool bVectorized) {
	int *start_d, *J_d;
	float *Val_d, *x_d, *y_d;

	/************************/
	/* copy to device       */
	/************************/

	hipMalloc((void **) &start_d, (N+1) * sizeof(int));
	checkCUDAError();
	hipMemcpy(start_d, start, (N+1) * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &J_d, nnz * sizeof(int));
	checkCUDAError();
	hipMemcpy(J_d, J, nnz * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &Val_d, nnz * sizeof(float));
	checkCUDAError();
	hipMemcpy(Val_d, Val, nnz * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &x_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &y_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(y_d, y, N * sizeof(float) , hipMemcpyHostToDevice);
	checkCUDAError();

	/************************/
	/* start kernel         */
	/************************/

	if (bVectorized) {
		//TODO: define grid and block size correctly
		dim3 grid(1, 1, N/TILE_SIZE);
		dim3 block(1, 1, TILE_SIZE);

		k_csr2_mat_vec_mm <<< grid, block >>> (start_d, J_d, Val_d, N, x_d, y_d);
	} else {
		dim3 grid((N - 1)/TILE_SIZE + 1, 1, 1);
		dim3 block(TILE_SIZE, 1, 1);

		k_csr_mat_vec_mm <<< grid, block >>> (start_d, J_d, Val_d, N, x_d, y_d);
	}

	checkCUDAError();

	/************************/
	/* copy back            */
	/************************/

	hipMemcpy(y, y_d, N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError();

	/************************/
	/* free memory          */
	/************************/
	hipFree(start_d);
	hipFree(J_d);
	hipFree(Val_d);
	hipFree(x_d);
	hipFree(y_d);
}

/**
 * Perform: ELL(A)x = y
 */
void ELLmatvecmult(int N, int num_cols_per_row , int * indices,
		float * data , float * x , float * y) {
	int *indices_d;
	float *data_d, *x_d, *y_d;

	/************************/
	/* copy to device       */
	/************************/

	hipMalloc((void **) &indices_d, N * num_cols_per_row * sizeof(int));
	checkCUDAError();
	hipMemcpy(indices_d, indices, N * num_cols_per_row * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &data_d, N * num_cols_per_row * sizeof(float));
	checkCUDAError();
	hipMemcpy(data_d, data, N * num_cols_per_row * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &x_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	hipMalloc((void **) &y_d, N * sizeof(float));
	checkCUDAError();
	hipMemcpy(y_d, y, N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError();

	/************************/
	/* start kernel         */
	/************************/

	//NYI: define grid and block size
	//k_ell_mat_vec_mm <<< grid, block >>> (N, num_cols_per_row, indices_d, data_d , x_d, y_d);
	checkCUDAError();

	/************************/
	/* copy back            */
	/************************/

	hipMemcpy(y, y_d, N * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError();

	/************************/
	/* free memory          */
	/************************/

	hipFree(indices_d);
	hipFree(data_d);
	hipFree(x_d);
	hipFree(y_d);
}

